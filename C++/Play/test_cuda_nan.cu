
#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_fp16.h>
#include<hip/hip_bf16.h>
#include<stdio.h>

__global__ void test_nan(){
    half _a = -1.0;
    half a = hsqrt(_a);
    __hip_bfloat16 bf = -1.0;
    __hip_bfloat16 bf2 = hsqrt(bf);
    float b = __fsqrt_rn(-1);
    double c = __dsqrt_rn(-1);
    printf("half sqrt(-1): %x\n", a);
    printf("bfloat16 sqrt(-1): %x\n", bf);
    printf("float sqrt(-1): %x\n", *(int*)(&b));
    printf("double sqrt(-1): %lx\n", *(long int*)(&c));
}

int main(){
    test_nan<<<1,1>>>();
    hipDeviceSynchronize();
}
