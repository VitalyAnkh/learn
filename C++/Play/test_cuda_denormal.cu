// test the behaviour of nvcc's `-ftz=true` option
// compile with: nvcc --std=c++17 --ftz=true -arch=sm_80 test_cuda_denormal.cu

#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_fp16.h>
#include<hip/hip_bf16.h>
#include<stdio.h>
#include<inttypes.h>

typedef unsigned char *byte_pointer;

// auxiliary function to print the bit sequence of datum
__device__ void show_bytes(unsigned char *start, int len) {
    int i;
    // little endian
    // print the bytes in hex
    printf("0x ");
    // use int other than size_t to avoid underflow
    // size_t is unsigned and (0u - 1) is very big
    for (i = len - 1; 0 <= i; i--) {
        printf("%.2x ", start[i]);
    }
    printf("\n");
}

__global__ void test_float_denormal(){
    // float f1 = 0.000000000000000000000000000000000000000000001;
    // float f2 = 0.000000000000000000000000000000000000000000002;
    float f3 = 0.000000000000000000000000000000000000000000003;
    float f4 = 0.000000000000000000000000000000000000000000004;
    float f5 = 0.000000000000000000000000000000000000000000005;

    uint32_t u1 = 0x8000'0001u;
    auto f1 = *(float *)(&u1);

    uint32_t u2 = 0x8000'0001u;
    auto f2 = *(float *)(&u2);

    float f6 = f1 + f2;
    printf("float f1: ");
    show_bytes((byte_pointer)&f1, sizeof(float));

    printf("float f2: ");
    show_bytes((byte_pointer)&f2, sizeof(float));

    printf("float f3: ");
    show_bytes((byte_pointer)&f3, sizeof(float));

    printf("float f4: ");
    show_bytes((byte_pointer)&f4, sizeof(float));

    printf("float f5: ");
    show_bytes((byte_pointer)&f5, sizeof(float));

    printf("float f6: ");
    show_bytes((byte_pointer)&f6, sizeof(float));

    printf("f1 in decimal: %.80f\n", f1);
    printf("f2 in decimal: %.80f\n", f2);
    printf("f6 in decimal: %.80f\n", f6);
}
__global__ void test_double_denormal(){
    uint64_t u1 = 0x0000'0000'0000'0001ull;
    auto f1 = *(double *)(&u1);
    uint64_t u2 = 0x0000'0000'0000'0002ull;
    auto f2 = *(double *)(&u2);
    uint64_t u3 = 0x0000'0000'0000'0003ull;
    auto f3 = *(double *)(&u3);
    uint64_t u4 = 0x0000'0000'0000'0004ull;
    auto f4 = *(double *)(&u4);
    uint64_t u5 = 0x0000'0000'0000'0005ull;
    auto f5 = *(double *)(&u5);

    double f6 = f1 + f2;

    printf("double f1: ");
    show_bytes((byte_pointer)&f1, sizeof(double));

    printf("double f2: ");
    show_bytes((byte_pointer)&f2, sizeof(double));

    printf("double f3: ");
    show_bytes((byte_pointer)&f3, sizeof(double));

    printf("double f4: ");
    show_bytes((byte_pointer)&f4, sizeof(double));

    printf("double f5: ");
    show_bytes((byte_pointer)&f5, sizeof(double));

    printf("double f6: ");
    show_bytes((byte_pointer)&f6, sizeof(double));

    printf("f1 in decimal: %.80lf\n", f1);
    printf("f2 in decimal: %.80lf\n", f2);
    printf("f6 in decimal: %.80lf\n", f6);
}

__global__ void test_half_denormal(){
    uint16_t u1 = 0x0000'0001u;
    auto f1 = *(half*)(&u1);
    uint16_t u2 = 0x0000'0002u;
    auto f2 = *(half*)(&u2);
    uint16_t u3 = 0x0000'0003u;
    auto f3 = *(half*)(&u3);
    uint16_t u4 = 0x0000'0004u;
    auto f4 = *(half*)(&u4);
    uint16_t u5 = 0x0000'0005u;
    auto f5 = *(half*)(&u5);

    half f6 = f1 + f2;
    printf("half f1: ");
    show_bytes((byte_pointer)&f1, sizeof(half));
    printf("half f2: ");
    show_bytes((byte_pointer)&f2, sizeof(half));
    printf("half f3: ");
    show_bytes((byte_pointer)&f3, sizeof(half));
    printf("half f4: ");
    show_bytes((byte_pointer)&f4, sizeof(half));
    printf("half f5: ");
    show_bytes((byte_pointer)&f5, sizeof(half));
    printf("half f6: ");
    show_bytes((byte_pointer)&f6, sizeof(half));
    printf("half f1 in decimal: %.50f\n", f1);
    printf("half f2 in decimal: %.50f\n", f2);
    printf("half f6 in decimal: %.50f\n", f6);
}

__global__ void test_bfloat16_denormal(){
    uint16_t u1 = 0x0000'0001u;
    auto f1 = *(__hip_bfloat16*)(&u1);

    uint16_t u2 = 0x0000'0002u;
    auto f2 = *(__hip_bfloat16*)(&u2);

    __hip_bfloat16 f3 = f1 + f2;

    printf("bfloat16 f1: ");
    show_bytes((byte_pointer)&f1, sizeof(__hip_bfloat16));
    printf("bfloat16 f2: ");
    show_bytes((byte_pointer)&f2, sizeof(__hip_bfloat16));
    printf("bfloat16 f3: ");
    show_bytes((byte_pointer)&f3, sizeof(__hip_bfloat16));
    printf("bfloat16 f1 in decimal: %.50f\n", f1);
    printf("bfloat16 f2 in decimal: %.50f\n", f2);
    printf("bfloat16 f3 in decimal: %.50f\n", f3);
}

int main(){
    test_float_denormal<<<1,1>>>();
    test_double_denormal<<<1,1>>>();
    test_half_denormal<<<1,1>>>();
    test_bfloat16_denormal<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
