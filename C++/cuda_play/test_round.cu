
#include <hip/hip_runtime.h>
#include <cstdio>

typedef unsigned char *byte_pointer;

__device__ void show_bytes(unsigned char *start, int len) {
    int i;
    // little endian
    // print the bytes in hex
    printf("Hex: 0x");
    // use int other than size_t to avoid underflow
    // size_t is unsigned and (0u - 1) is very big
    for (i = len - 1; 0 <= i; i--) {
        printf("%.2x ", start[i]);
    }
    printf("\n");
}

__global__ void test_double_type_cast(double a) {
    printf("the number's original value is: %.18f\n", a);
    printf("the number's original bits is: ");
    show_bytes((unsigned char *) (&a), sizeof(double));

    printf("cuda __double2float_rd() is: ");
    float b = __double2float_rd(*((double *) &a));
    show_bytes((unsigned char *) (&b), sizeof(float));

    b = __double2float_rn(*((double *) &a));
    printf("cuda __double2float_rn() is: ");
    show_bytes((unsigned char *) (&b), sizeof(float));

    printf("cuda __double2float_ru() is: ");
    b = __double2float_ru(*((double *) &a));
    show_bytes((unsigned char *) (&b), sizeof(float));

    printf("cuda __double2float_rz() is: ");
    b = __double2float_rz(*((double *) &a));
    show_bytes((unsigned char *) (&b), sizeof(float));

    printf("cuda __double2hiint() is: ");
    int c = __double2hiint(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_rd() is: ");
    c = __double2int_rd(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_rn() is: ");
    c = __double2int_rn(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_ru() is: ");
    c = __double2int_ru(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_rz() is: ");
    c = __double2int_rz(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2ll_rd() is: ");
    long long d_long_long_1 = __double2ll_rd(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));

    printf("cuda __double2ll_rn() is: ");
    d_long_long_1 = __double2ll_rn(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));

    printf("cuda __double2ll_ru() is: ");
    d_long_long_1 = __double2ll_ru(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));

    printf("cuda __double2ll_rz() is: ");
    d_long_long_1 = __double2ll_rz(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));


    printf("cuda __double2loint() is: ");
    int d_int = __double2loint(a);
    show_bytes((unsigned char *) (&d_int), sizeof(long long));

    printf("cuda __double2ll_rd() is: ");
    long long d_long_long = __double2ll_rd(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2ll_rn() is: ");
    d_long_long = __double2ll_rn(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2ll_ru() is: ");
    d_long_long = __double2ll_ru(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2ll_rz() is: ");
    d_long_long = __double2ll_rz(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2uint_rd() is: ");
    unsigned int d_unsigned_int = __double2uint_rd(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2uint_rn() is: ");
    d_unsigned_int = __double2uint_rn(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2uint_ru() is: ");
    d_unsigned_int = __double2uint_ru(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2uint_rz() is: ");
    d_unsigned_int = __double2uint_rz(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2ull_rd() is: ");
    unsigned long long d_unsigned_long_long = __double2ull_rd(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("cuda __double2ull_rn() is: ");
    d_unsigned_long_long = __double2ull_rn(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("cuda __double2ull_ru() is: ");
    d_unsigned_long_long = __double2ull_ru(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("cuda __double2ull_rz() is: ");
    d_unsigned_long_long = __double2ull_rz(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("===================================================================\n");
}

__global__ void test_float_type_cast(float b) {
    float a = b;
    printf("cuda __float2int_rd() is: ");
    int c = __float2int_rd(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rn() is: ");
    c = __float2int_rn(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_ru() is: ");
    c = __float2int_ru(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rz() is: ");
    c = __float2int_rz(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2uint_rd() is: ");
    unsigned int d = __float2uint_rd(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rn() is: ");
    d = __float2uint_rn(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_ru() is: ");
    d = __float2uint_ru(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rz() is: ");
    d = __float2uint_rz(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2ll_rd() is: ");
    long long e = __float2ll_rd(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rn() is: ");
    e = __float2ll_rn(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_ru() is: ");
    e = __float2ll_ru(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rz() is: ");
    e = __float2ll_rz(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ull_rd() is: ");
    unsigned long long f = __float2ull_rd(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rn() is: ");
    f = __float2ull_rn(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_ru() is: ");
    f = __float2ull_ru(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rz() is: ");
    f = __float2ull_rz(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2int_rz() is: ");
    c = __float2int_rz(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_ru() is: ");
    c = __float2int_ru(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rn() is: ");
    c = __float2int_rn(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rd() is: ");
    c = __float2int_rd(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2uint_rz() is: ");
    d = __float2uint_rz(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_ru() is: ");
    d = __float2uint_ru(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rn() is: ");
    d = __float2uint_rn(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rd() is: ");
    d = __float2uint_rd(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2ll_rz() is: ");
    e = __float2ll_rz(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_ru() is: ");
    e = __float2ll_ru(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rn() is: ");
    e = __float2ll_rn(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rd() is: ");
    e = __float2ll_rd(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ull_rz() is: ");
    f = __float2ull_rz(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_ru() is: ");
    f = __float2ull_ru(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rn() is: ");
    f = __float2ull_rn(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rd() is: ");
    f = __float2ull_rd(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2int_rz() is: ");
    c = __float2int_rz(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_ru() is: ");
    c = __float2int_ru(b);
    show_bytes((unsigned char *) (&c), sizeof(int));
    printf("===================================================================\n");
}

__global__ void test_int_type_cast(int a) {

    printf("the number's original value is: %d\n", a);
    printf("the number's original bits is: ");
    show_bytes((unsigned char *) (&a), sizeof(int));

    printf("cuda __int2float_rn() is: ");
    float b_float = __int2float_rn(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2float_rd() is: ");
    b_float = __int2float_rd(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2float_ru() is: ");
    b_float = __int2float_ru(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2float_rz() is: ");
    b_float = __int2float_rz(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2double_rn() is: ");
    double b_double = __int2double_rn(a);
    show_bytes((unsigned char *) (&b_double), sizeof(double));

    printf("cuda __int_as_float() is: ");
    b_float = __int_as_float(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));
    printf("=========================================================\n");
}

int main() {
    test_double_type_cast<<<1, 1>>>(0.123456789123456789);
    test_double_type_cast<<<1,1>>>(123.123456789123456789);
    test_float_type_cast<<<1, 1>>>(0.123456789123456789);
    test_float_type_cast<<<1,1>>>(123.123456789123456789);
    test_int_type_cast<<<1, 1>>>(2);
    hipDeviceSynchronize();
    return 0;
}