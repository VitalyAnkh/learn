
#include <hip/hip_runtime.h>
#include <cstdio>

typedef unsigned char *byte_pointer;

__device__ void show_bytes(unsigned char *start, int len) {
    int i;
    // little endian
    // print the bytes in hex
    printf("Hex: 0x");
    // use int other than size_t to avoid underflow
    // size_t is unsigned and (0u - 1) is very big
    for (i = len - 1; 0 <= i; i--) {
        printf("%.2x ", start[i]);
    }
    printf("\n");
}

__global__ void test_double_type_cast(double a) {
    printf("the number's original value is: %.18f\n", a);
    printf("the number's original bits is: ");
    show_bytes((unsigned char *) (&a), sizeof(double));

    printf("cuda __double2float_rd() is: ");
    float b = __double2float_rd(*((double *) &a));
    show_bytes((unsigned char *) (&b), sizeof(float));

    b = __double2float_rn(*((double *) &a));
    printf("cuda __double2float_rn() is: ");
    show_bytes((unsigned char *) (&b), sizeof(float));

    printf("cuda __double2float_ru() is: ");
    b = __double2float_ru(*((double *) &a));
    show_bytes((unsigned char *) (&b), sizeof(float));

    printf("cuda __double2float_rz() is: ");
    b = __double2float_rz(*((double *) &a));
    show_bytes((unsigned char *) (&b), sizeof(float));

    printf("cuda __double2hiint() is: ");
    int c = __double2hiint(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_rd() is: ");
    c = __double2int_rd(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_rn() is: ");
    c = __double2int_rn(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_ru() is: ");
    c = __double2int_ru(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2int_rz() is: ");
    c = __double2int_rz(a);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __double2ll_rd() is: ");
    long long d_long_long_1 = __double2ll_rd(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));

    printf("cuda __double2ll_rn() is: ");
    d_long_long_1 = __double2ll_rn(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));

    printf("cuda __double2ll_ru() is: ");
    d_long_long_1 = __double2ll_ru(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));

    printf("cuda __double2ll_rz() is: ");
    d_long_long_1 = __double2ll_rz(a);
    show_bytes((unsigned char *) (&d_long_long_1), sizeof(long long));


    printf("cuda __double2loint() is: ");
    int d_int = __double2loint(a);
    show_bytes((unsigned char *) (&d_int), sizeof(long long));

    printf("cuda __double2ll_rd() is: ");
    long long d_long_long = __double2ll_rd(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2ll_rn() is: ");
    d_long_long = __double2ll_rn(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2ll_ru() is: ");
    d_long_long = __double2ll_ru(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2ll_rz() is: ");
    d_long_long = __double2ll_rz(a);
    show_bytes((unsigned char *) (&d_long_long), sizeof(long long));

    printf("cuda __double2uint_rd() is: ");
    unsigned int d_unsigned_int = __double2uint_rd(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2uint_rn() is: ");
    d_unsigned_int = __double2uint_rn(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2uint_ru() is: ");
    d_unsigned_int = __double2uint_ru(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2uint_rz() is: ");
    d_unsigned_int = __double2uint_rz(a);
    show_bytes((unsigned char *) (&d_unsigned_int), sizeof(unsigned int));

    printf("cuda __double2ull_rd() is: ");
    unsigned long long d_unsigned_long_long = __double2ull_rd(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("cuda __double2ull_rn() is: ");
    d_unsigned_long_long = __double2ull_rn(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("cuda __double2ull_ru() is: ");
    d_unsigned_long_long = __double2ull_ru(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("cuda __double2ull_rz() is: ");
    d_unsigned_long_long = __double2ull_rz(a);
    show_bytes((unsigned char *) (&d_unsigned_long_long), sizeof(unsigned long long));

    printf("===================================================================\n");
}

__global__ void test_float_type_cast(float b) {
    float a = b;
    printf("cuda __float2int_rd() is: ");
    int c = __float2int_rd(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rn() is: ");
    c = __float2int_rn(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_ru() is: ");
    c = __float2int_ru(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rz() is: ");
    c = __float2int_rz(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2uint_rd() is: ");
    unsigned int d = __float2uint_rd(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rn() is: ");
    d = __float2uint_rn(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_ru() is: ");
    d = __float2uint_ru(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rz() is: ");
    d = __float2uint_rz(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2ll_rd() is: ");
    long long e = __float2ll_rd(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rn() is: ");
    e = __float2ll_rn(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_ru() is: ");
    e = __float2ll_ru(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rz() is: ");
    e = __float2ll_rz(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ull_rd() is: ");
    unsigned long long f = __float2ull_rd(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rn() is: ");
    f = __float2ull_rn(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_ru() is: ");
    f = __float2ull_ru(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rz() is: ");
    f = __float2ull_rz(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2int_rz() is: ");
    c = __float2int_rz(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_ru() is: ");
    c = __float2int_ru(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rn() is: ");
    c = __float2int_rn(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_rd() is: ");
    c = __float2int_rd(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2uint_rz() is: ");
    d = __float2uint_rz(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_ru() is: ");
    d = __float2uint_ru(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rn() is: ");
    d = __float2uint_rn(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2uint_rd() is: ");
    d = __float2uint_rd(b);
    show_bytes((unsigned char *) (&d), sizeof(unsigned int));

    printf("cuda __float2ll_rz() is: ");
    e = __float2ll_rz(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_ru() is: ");
    e = __float2ll_ru(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rn() is: ");
    e = __float2ll_rn(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ll_rd() is: ");
    e = __float2ll_rd(b);
    show_bytes((unsigned char *) (&e), sizeof(long long));

    printf("cuda __float2ull_rz() is: ");
    f = __float2ull_rz(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_ru() is: ");
    f = __float2ull_ru(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rn() is: ");
    f = __float2ull_rn(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2ull_rd() is: ");
    f = __float2ull_rd(b);
    show_bytes((unsigned char *) (&f), sizeof(unsigned long long));

    printf("cuda __float2int_rz() is: ");
    c = __float2int_rz(b);
    show_bytes((unsigned char *) (&c), sizeof(int));

    printf("cuda __float2int_ru() is: ");
    c = __float2int_ru(b);
    show_bytes((unsigned char *) (&c), sizeof(int));
    printf("===================================================================\n");
}

__global__ void test_int_type_cast(int a) {

    printf("the number's original value is: %d\n", a);
    printf("the number's original bits is: ");
    show_bytes((unsigned char *) (&a), sizeof(int));

    printf("cuda __int2float_rn() is: ");
    float b_float = __int2float_rn(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2float_rd() is: ");
    b_float = __int2float_rd(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2float_ru() is: ");
    b_float = __int2float_ru(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2float_rz() is: ");
    b_float = __int2float_rz(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    printf("cuda __int2double_rn() is: ");
    double b_double = __int2double_rn(a);
    show_bytes((unsigned char *) (&b_double), sizeof(double));

    printf("cuda __int_as_float() is: ");
    b_float = __int_as_float(a);
    show_bytes((unsigned char *) (&b_float), sizeof(float));

    float b_float_2 = -1.5;
    printf("original value is %f\n", b_float_2);
    printf("original bit pattern is: ");
    show_bytes((unsigned char *) (&b_float_2), sizeof(float));

    int c_int = __float2int_rn(b_float_2);
    printf("cuda __float2int_rn(): value: %d, ", c_int);
    show_bytes((unsigned char *) (&c_int), sizeof(int));

    c_int = __float2int_ru(b_float_2);
    printf("cuda __float2int_ru(): value: %d, ", c_int);
    show_bytes((unsigned char *) (&c_int), sizeof(int));

    c_int = __float2int_rd(b_float_2);
    printf("cuda __float2int_rd(): value: %d, ", c_int);
    show_bytes((unsigned char *) (&c_int), sizeof(int));

    c_int = __float2int_rz(b_float_2);
    printf("cuda __float2int_rz(): value: %d, ", c_int);
    show_bytes((unsigned char *) (&c_int), sizeof(int));

    int d_longlong = 123456789;
    printf("original value is %lld\n", d_longlong);
    printf("original bit pattern is: ");
    show_bytes((unsigned char *) (&d_longlong), sizeof(long long));
    float e_float = __int2float_rn(d_longlong);
    printf("cuda __int2float_rn(): value: %f, \n", e_float);

    e_float = __int2float_ru(d_longlong);
    printf("cuda __int2float_ru(): value: %f, \n", e_float);

    e_float = __int2float_rd(d_longlong);
    printf("cuda __int2float_rd(): value: %f, \n", e_float);

    e_float = __int2float_rz(d_longlong);
    printf("cuda __int2float_rz(): value: %f, \n", e_float);
    printf("=========================================================\n");

    float sample_nan = 0.0/0.0;
    unsigned long long ull = __float2ull_rn(sample_nan);
    show_bytes((unsigned char *)&ull, sizeof(unsigned long long));
    printf("TEST NAN cuda __float2ull_rn(): value: %llx, \n", ull);
    printf("=========================================================\n");

    unsigned long long ui = __float2uint_rn(sample_nan);
    show_bytes((unsigned char *)&ui, sizeof(unsigned long long));
    printf("TEST NAN cuda __float2uint_rn(): value: %llx, \n", ui);
    printf("=========================================================\n");
}

int main() {
    test_double_type_cast<<<1, 1>>>(0.123456789123456789);
    test_double_type_cast<<<1, 1>>>(123.123456789123456789);
    test_float_type_cast<<<1, 1>>>(0.123456789123456789);
    test_float_type_cast<<<1, 1>>>(123.123456789123456789);
    test_int_type_cast<<<1, 1>>>(2);
    hipDeviceSynchronize();
    return 0;
}