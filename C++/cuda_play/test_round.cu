
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void test_double_type_cast(double a) {
    printf("cuda __double2float_rd(%.18f) is: %f\n", a, __double2float_rd(a));
    printf("cuda __double2float_rn(%.18f) is: %f\n", a, __double2float_rn(a));
    printf("cuda __double2float_ru(%.18f) is: %f\n", a, __double2float_ru(a));
    printf("cuda __double2float_rz(%.18f) is: %f\n", a, __double2float_rz(a));
    printf("cuda __double2hiint(%e) is: %d\n", a, __double2hiint(a));
    printf("cuda __double2int_rd(%f) is: %d\n", a, __double2int_rd(a));
    printf("cuda __double2int_rn(%f) is: %d\n", a, __double2int_rn(a));
    printf("cuda __double2int_ru(%f) is: %d\n", a, __double2int_ru(a));
    printf("cuda __double2int_rz(%f) is: %d\n", a, __double2int_rz(a));
    printf("cuda __double2ll_rd(%f) is: %lld\n", a, __double2ll_rd(a));
    printf("cuda __double2ll_rn(%f) is: %lld\n", a, __double2ll_rn(a));
    printf("cuda __double2ll_ru(%f) is: %lld\n", a, __double2ll_ru(a));
    printf("cuda __double2ll_rz(%f) is: %lld\n", a, __double2ll_rz(a));
    printf("cuda __double2loint(%f) is: %d\n", a, __double2loint(a));
    printf("cuda __double2uint_rd(%f) is: %d\n", a, __double2uint_rd(a));
    printf("cuda __double2uint_rn(%f) is: %d\n", a, __double2uint_rn(a));
    printf("cuda __double2uint_ru(%f) is: %d\n", a, __double2uint_ru(a));
    printf("cuda __double2uint_rz(%f) is: %d\n", a, __double2uint_rz(a));
    printf("cuda __double2ull_rd(%f) is: %lld\n", a, __double2ull_rd(a));
    printf("cuda __double2ull_rn(%f) is: %lld\n", a, __double2ull_rn(a));
    printf("cuda __double2ull_ru(%f) is: %lld\n", a, __double2ull_ru(a));
    printf("cuda __double2ull_rz(%f) is: %lld\n", a, __double2ull_rz(a));
    printf("cuda __double_as_longlong(%f) is: %lld\n", a, __double_as_longlong(a));
    printf("=================================================================\n");
}

__global__ void test_float_type_cast(float a) {
    printf("cuda __float2int_rd(%f) is: %d\n", a, __float2int_rd(a));
    printf("cuda __float2int_rn(%f) is: %d\n", a, __float2int_rn(a));
    printf("cuda __float2int_ru(%f) is: %d\n", a, __float2int_ru(a));
    printf("cuda __float2int_rz(%f) is: %d\n", a, __float2int_rz(a));
    printf("cuda __float2ll_rd(%f) is: %lld\n", a, __float2ll_rd(a));
    printf("cuda __float2ll_rn(%f) is: %lld\n", a, __float2ll_rn(a));
    printf("cuda __float2ll_ru(%f) is: %lld\n", a, __float2ll_ru(a));
    printf("cuda __float2ll_rz(%f) is: %lld\n", a, __float2ll_rz(a));
    printf("cuda __float2uint_rd(%f) is: %d\n", a, __float2uint_rd(a));
    printf("cuda __float2uint_rn(%f) is: %d\n", a, __float2uint_rn(a));
    printf("===================================================================\n");
}

__global__ void test_int_type_cast(int a) {
    printf("cuda __int2double_rn(%d) is: %f\n", a, __int2double_rn(a));
    printf("cuda __int2float_rd(%d) is: %f\n", a, __int2float_rd(a));
    printf("cuda __int2float_rn(%d) is: %f\n", a, __int2float_rn(a));
    printf("cuda __int2float_ru(%d) is: %f\n", a, __int2float_ru(a));
    printf("cuda __int2float_rz(%d) is: %f\n", a, __int2float_rz(a));
    printf("cuda __int_as_float(%d) is: %f\n", a, __int_as_float(a));
    printf("=========================================================\n");
}

int main() {
    test_double_type_cast<<<1, 1>>>(1.40);
    test_double_type_cast<<<1, 1>>>(0.123456789123456789);
    test_float_type_cast<<<1, 1>>>(0.123456789123456789);
    test_int_type_cast<<<1, 1>>>(2);
    hipDeviceSynchronize();
    return 0;
}