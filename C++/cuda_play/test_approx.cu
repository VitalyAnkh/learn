#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hip/hip_bf16.h>

__global__ void test_approx(float f) {

    float sqrt = sqrtf(f);
    float div = f /sqrt;
    float tan_approx = tanf(f);
    double sqrt_d = sqrtf(f);

    // printf("%f\n", sqrt(4.5));
    printf( "div: %f\n", div);
    printf("tan_approx: %f\n", tan_approx);
    printf("sqrt_d: %f\n", sqrt_d);
}

// use this function to test nvcc --ftz=true behaviour
__global__ void test_ftz(float f){
    float small = 8.668306183451529e-40 ;
    printf("small: %f\n", small);
    printf("small: %a\n", small);
    f = f + small;
    printf("f: %f\n", f);
    printf("f: %a\n", f);
    printf("f + small: %2a\n", f);
    float a = 0.00000114514;
    a = a + f;
    printf("a + f: %f\n", a);
}

int main(){
  test_approx<<<1,1>>>(4.5);
  test_ftz<<<1,1>>>(4.5);
  hipDeviceSynchronize();
  float i = INFINITY;
  printf("INFINITY: %f\n", i);
}
