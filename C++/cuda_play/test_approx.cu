#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hip/hip_bf16.h>

__global__ void test_approx(float f) {

    float sqrt = sqrtf(f);
    float div = f /sqrt;
    float tan_approx = tanf(f);
    double sqrt_d = sqrtf(f);

    // printf("%f\n", sqrt(4.5));
    printf( "div: %f\n", div);
    printf("tan_approx: %f\n", tan_approx);
    printf("sqrt_d: %f\n", sqrt_d);
}

int main(){
  test_approx<<<1,1>>>(4.5);
  hipDeviceSynchronize();
}
