#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void add_sync(){
    int  a = 1 ;
    int b = __reduce_add_sync(0xffffffff, a);
    printf("%d\n", threadIdx.x + b);
    __syncthreads();
    printf("%d\n", b);
}

int main(){
    add_sync<<<1,64>>>();
    hipDeviceSynchronize();
}