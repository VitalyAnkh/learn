#include <hip/hip_runtime.h>
#include<iostream>

__global__ void vec_add_kernel(float *a, float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) { c[i] = a[i] + b[i]; }
}

void vec_add(float *a, float *b, float *c, int n) {
    float *d_a, *d_b, *d_c;
    int size = n * sizeof(float);
    hipMalloc((void **) &d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_c, size);

    vec_add_kernel<<<ceil(n / 256.0), 256>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}


int main() {

    int const N = 1000;
    float a[N];
    float b[N];
    float c[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i + 1;
    }

    vec_add(a, b, c, N);
    for (int i = 0; i < N; i++) {
        std::cout << c[i] << " ";
    }
    return 0;
}