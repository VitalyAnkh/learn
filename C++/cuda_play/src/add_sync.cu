#include <hip/hip_runtime.h>

#include <stdio.h>

// _Float32 f;
__global__ void add_sync() {
    int a = 1;
    int b = 2;
    __reduce_add_sync(0xffffffff, a);
    printf("%d\n", threadIdx.x + b);
    __syncthreads();
    printf("%d\n", b);
}

int main() {
    add_sync<<<1, 64>>>();
    hipDeviceSynchronize();
}
