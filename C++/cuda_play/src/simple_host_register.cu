#include <hip/hip_runtime.h>
#include <stdio.h>

// 简单的核函数：将数组元素乘2
__global__ void doubleElements(float *array, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    array[idx] = 2.0f * array[idx];
  }
}

int main() {
  const int N = 1024; // 更小的数组大小
  float *host_array = new float[N];
  // 初始化数组
  for (int i = 0; i < N; i++) {
    host_array[i] = static_cast<float>(i);
  }

  printf("Before kernel run: ===========\n");
  // 验证结果
  printf("First element: %f\n", host_array[0]);
  printf("Second element: %f\n", host_array[1]);
  printf("Last element: %f\n", host_array[N - 1]);
  // 注册主机内存
  hipHostRegister(host_array, N * sizeof(float), hipHostRegisterMapped);
  // 获取GPU可访问的指针
  float *device_pointer;
  hipHostGetDevicePointer(&device_pointer, host_array, 0);

  // 运行核函数
  // doubleElements<<<4, 256>>>(device_pointer, N);
  // or use the host address
  doubleElements<<<4, 256>>>(host_array, N);

  hipDeviceSynchronize();

  printf("After run: ===========\n");
  // 验证结果
  printf("First element: %f\n", host_array[0]);
  printf("Second element: %f\n", host_array[1]);
  printf("Last element: %f\n", host_array[N - 1]);

  // 清理
  hipHostUnregister(host_array);
  delete[] host_array;

  return 0;
}
