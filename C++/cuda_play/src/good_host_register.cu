#include <hip/hip_runtime.h>
#include <stdio.h>

// 错误检查宏
#define CHECK_CUDA_ERROR(call)                                                 \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// CUDA核函数：将数组中的每个元素乘2
__global__ void doubleElements(float *dev_array, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    dev_array[idx] = 2.0f * dev_array[idx];
  }
}

int main() {
  const int N = 1000000; // 数组大小
  const size_t size = N * sizeof(float);

  // 在主机端分配普通内存
  float *host_array = new float[N];

  // 初始化数组
  for (int i = 0; i < N; i++) {
    host_array[i] = static_cast<float>(i);
  }

  // 将主机内存注册为固定内存
  CHECK_CUDA_ERROR(hipHostRegister(host_array, size, hipHostRegisterDefault));

  // 分配设备内存
  float *dev_array;
  CHECK_CUDA_ERROR(hipMalloc(&dev_array, size));

  // 记录时间
  hipEvent_t start, stop;
  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  // 开始计时
  CHECK_CUDA_ERROR(hipEventRecord(start));

  // 将数据从主机复制到设备
  CHECK_CUDA_ERROR(
      hipMemcpy(dev_array, host_array, size, hipMemcpyHostToDevice));

  // 启动核函数
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  doubleElements<<<blocksPerGrid, threadsPerBlock>>>(dev_array, N);

  // 检查核函数执行错误
  CHECK_CUDA_ERROR(hipGetLastError());

  // 将结果复制回主机
  CHECK_CUDA_ERROR(
      hipMemcpy(host_array, dev_array, size, hipMemcpyDeviceToHost));

  // 停止计时
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));

  // 计算耗时
  float milliseconds = 0;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Total time: %f ms\n", milliseconds);

  // 验证结果
  bool success = true;
  for (int i = 0; i < N; i++) {
    if (host_array[i] != static_cast<float>(i) * 2.0f) {
      printf("Verification failed at index %d\n", i);
      success = false;
      break;
    }
  }
  if (success) {
    printf("Verification passed!\n");
  }

  // 清理资源
  CHECK_CUDA_ERROR(hipHostUnregister(host_array)); // 取消注册固定内存
  CHECK_CUDA_ERROR(hipFree(dev_array));
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));

  delete[] host_array;

  return 0;
}
