#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void foo_kernel(float *a, float *b, unsigned int N) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    b[i] = 2.7f * a[i] - 4.3f;
  }
  printf("%f\n", b);

  __syncthreads();
}

int main() {
  unsigned int N = 20000;
  hipDeviceSynchronize();
}
