#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(call)                                                 \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// CUDA核函数：直接操作主机内存
__global__ void doubleElements(float *array, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    // 直接读写固定内存
    array[idx] = 2.0f * array[idx];
  }
}

int main() {
  const int N = 1000000;
  const size_t size = N * sizeof(float);

  // 分配主机内存
  float *host_array = new float[N];

  // 初始化数组
  for (int i = 0; i < N; i++) {
    host_array[i] = static_cast<float>(i);
  }

  // 注册主机内存，使其可以被GPU直接访问
  // cudaHostRegisterMapped 标志允许创建可由GPU直接访问的固定内存映射
  CHECK_CUDA_ERROR(hipHostRegister(
      host_array, size, hipHostRegisterMapped | hipHostRegisterPortable));

  // 获取映射到GPU的指针
  float *device_pointer;
  CHECK_CUDA_ERROR(hipHostGetDevicePointer(&device_pointer, host_array, 0));

  // 记录时间
  hipEvent_t start, stop;
  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  // 开始计时
  CHECK_CUDA_ERROR(hipEventRecord(start));

  // 启动核函数，直接使用映射的指针
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  doubleElements<<<blocksPerGrid, threadsPerBlock>>>(device_pointer, N);

  // 同步设备以确保计算完成
  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  // 停止计时
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));

  // 计算耗时
  float milliseconds = 0;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Total time: %f ms\n", milliseconds);

  // 验证结果 - 直接访问主机内存，因为它已经包含了结果
  bool success = true;
  for (int i = 0; i < N; i++) {
    if (host_array[i] != static_cast<float>(i) * 2.0f) {
      printf("Verification failed at index %d: expected %f, got %f\n", i,
             static_cast<float>(i) * 2.0f, host_array[i]);
      success = false;
      break;
    }
  }
  if (success) {
    printf("Verification passed!\n");
  }

  // 清理资源
  CHECK_CUDA_ERROR(hipHostUnregister(host_array));
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));

  delete[] host_array;

  return 0;
}
