#include <hip/hip_fp16.h>
#include <bitset>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <vector> // 添加此头文件以使用 std::vector

// 辅助函数：打印fp16的二进制表示并返回格式化的字符串
std::string getFp16Binary(const __half &h) {
  unsigned short bits;
  memcpy(&bits, &h, sizeof(__half));
  std::bitset<16> binary(bits);
  std::stringstream ss;
  ss << binary << " (s:" << binary[15]
     << " e:" << binary.to_string().substr(1, 5)
     << " f:" << binary.to_string().substr(6, 10) << ")";
  return ss.str();
}

// 辅助函数：打印fp8的二进制表示并返回格式化的字符串
std::string getFp8Binary(const __hip_fp8_storage_t &f) {
  unsigned char bits;
  memcpy(&bits, &f, sizeof(__hip_fp8_storage_t));
  std::bitset<8> binary(bits);
  std::stringstream ss;
  ss << binary << " (s:" << binary[7]
     << " e:" << binary.to_string().substr(4, 3)
     << " f:" << binary.to_string().substr(0, 4) << ")";
  return ss.str();
}

// CUDA kernel进行FP16到FP8的转换
__global__ void convertFp16ToFp8(const __half *input,
                                 __hip_fp8_storage_t *output, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;

  __half_raw raw_val = __half_raw(input[idx]);
  output[idx] =
      __hip_cvt_halfraw_to_fp8(raw_val, __hip_saturation_t::__HIP_SATFINITE,
                              __hip_fp8_interpretation_t::__HIP_E4M3_FNUZ);
}

// CUDA kernel进行FP8到FP16的转换
__global__ void convertFp8ToFp16(const __hip_fp8_storage_t *input, half *output,
                                 int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;

  __half_raw h =
      __hip_cvt_fp8_to_halfraw(input[idx], __hip_fp8_interpretation_t::__HIP_E4M3_FNUZ);
  output[idx] = __half(h);
}

int main() {
  const int size = 16;

  // 准备一些具有代表性的FP16值
  std::vector<__half> test_fp16_values = {
      __float2half(0.0f),    // 零
      __float2half(1.0f),    // 整数
      __float2half(-1.0f),   // 负数
      __float2half(0.5f),    // 1/2
      __float2half(-0.5f),   // -1/2
      __float2half(0.25f),   // 1/4
      __float2half(-0.25f),  // -1/4
      __float2half(0.125f),  // 1/8
      __float2half(2.0f),    // 2
      __float2half(-2.0f),   // -2
      __float2half(4.0f),    // 4
      __float2half(-4.0f),   // -4
      __float2half(0.0625f), // 1/16
      __float2half(8.0f),    // 8
      __float2half(0.333f),  // 非2的幂
      __float2half(-0.333f)  // 负的非2的幂
  };

  // 分配主机内存
  __half *h_input = new __half[size];
  __hip_fp8_storage_t *h_output_fp8 = new __hip_fp8_storage_t[size];
  __half *h_output_fp16 = new __half[size];

  // 复制测试值
  for (int i = 0; i < size; i++) {
    h_input[i] = test_fp16_values[i];
  }

  // 分配设备内存
  __half *d_input;
  __hip_fp8_storage_t *d_output_fp8;
  __half *d_output_fp16;
  hipMalloc(&d_input, size * sizeof(__half));
  hipMalloc(&d_output_fp8, size * sizeof(__hip_fp8_storage_t));
  hipMalloc(&d_output_fp16, size * sizeof(__half));

  // 复制数据到设备
  hipMemcpy(d_input, h_input, size * sizeof(__half), hipMemcpyHostToDevice);

  // 执行转换
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  convertFp16ToFp8<<<numBlocks, blockSize>>>(d_input, d_output_fp8, size);
  convertFp8ToFp16<<<numBlocks, blockSize>>>(d_output_fp8, d_output_fp16, size);

  // 复制结果回主机
  hipMemcpy(h_output_fp8, d_output_fp8, size * sizeof(__hip_fp8_storage_t),
             hipMemcpyDeviceToHost);
  hipMemcpy(h_output_fp16, d_output_fp16, size * sizeof(__half),
             hipMemcpyDeviceToHost);

  // 打印结果
  std::cout << "\nFP16 to FP8 Conversion Analysis:\n";
  std::cout << std::string(120, '-') << "\n";
  std::cout << "Format comparison: (s:sign e:exponent f:fraction)\n\n";

  // 表头
  std::cout << std::left << std::setw(15) << "Original(double)" << std::setw(15)
            << "Via FP8(double)" << std::setw(15) << "Abs Diff"
            << "| " << std::setw(45) << "FP16 Binary"
            << "| " << "FP8 Binary\n";
  std::cout << std::string(120, '-') << "\n";

  // 打印每个值的详细信息
  for (int i = 0; i < size; i++) {
    // 转换为double进行比较
    double orig_val = __half2float(h_input[i]);
    double converted_val = __half2float(h_output_fp16[i]);
    double abs_diff = std::abs(orig_val - converted_val);

    // 打印数值比较（使用double精度）
    std::cout << std::fixed << std::setprecision(6) << std::left
              << std::setw(15) << orig_val << std::setw(15) << converted_val
              << std::setw(15) << abs_diff << "| " << std::setw(45)
              << getFp16Binary(h_input[i]) << "| "
              << getFp8Binary(h_output_fp8[i]) << "\n";

    // 如果差异显著，添加警告
    if (abs_diff > 0.01 && orig_val != 0.0) {
      double rel_error = (abs_diff / std::abs(orig_val)) * 100;
      std::cout << "   ^-- Notable difference! Relative error: "
                << std::setprecision(2) << rel_error << "%\n";
    }
  }

  // 清理内存
  delete[] h_input;
  delete[] h_output_fp8;
  delete[] h_output_fp16;
  hipFree(d_input);
  hipFree(d_output_fp8);
  hipFree(d_output_fp16);

  return 0;
}
