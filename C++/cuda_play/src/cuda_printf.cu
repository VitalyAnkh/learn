#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test_printf() {
  printf("%d\n", 1);
  if (threadIdx.x % 2 == 0) {
    printf("%d\n", 2);
  }
  printf("%d\n", 3);
}

int main() {
  test_printf<<<1, 10>>>();
  hipDeviceSynchronize();
}
