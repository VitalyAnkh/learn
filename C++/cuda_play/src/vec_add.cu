
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
  __shared__ alignas(alignof(half)) half h;
__global__ void vec_add_kernel(float *a, float *b, float *c, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

void vec_add(float *a, float *b, float *c, int n) {
  float *d_a, *d_b, *d_c;
  int size = n * sizeof(float);
  hipMalloc((void **)&d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_b, size);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_c, size);

  vec_add_kernel<<<ceil(n / 256.0), 256>>>(d_a, d_b, d_c, n);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  int foo, bar;
  foo = 1;
  bar = 2;

  hipMemcpy(&foo, &bar, 1 * sizeof(int), hipMemcpyHostToDevice);
  printf("foo: %d, bar: %d\n", foo, bar);
}

int main() {
  int const N = 64;
  float a[N];
  float b[N];
  float c[N];
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = 2 * i + 1;
  }
  printf("===============\n");
  printf("a is: \n");
  for (int i = 0; i < N; i++) {
    printf("%f ", a[i]);
  }
  printf("===============\n");
  printf("b is: \n");
  for (int i = 0; i < N; i++) {
    printf("%f ", b[i]);
  }
  vec_add(a, b, c, N);

  printf("===============\n");
  printf("c is: \n");
  for (int i = 0; i < N; i++) {
    printf("%f ", c[i]);
  }

  return 0;
}
