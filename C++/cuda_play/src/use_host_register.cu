#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  // 定义主机侧的数据
  const size_t size = 1024 * 1024; // 1 MB 大小
  float *hostData;

  // 分配主机内存
  hostData = (float *)malloc(size * sizeof(float));
  if (hostData == nullptr) {
    std::cerr << "无法分配主机内存!" << std::endl;
    return -1;
  }

  // 用于将内存锁定，以便 CUDA 可以高效访问
  hipError_t err =
      hipHostRegister(hostData, size * sizeof(float), hipHostRegisterDefault);
  if (err != hipSuccess) {
    std::cerr << "hipHostRegister 失败: " << hipGetErrorString(err)
              << std::endl;
    free(hostData);
    return -1;
  }

  // 可以在这里执行内存传输，主机内存已锁定，因此可以更高效地传输数据
  // 使用 cudaMemcpy 或 cudaMemcpyAsync 在 GPU 和 CPU 之间传递数据

  // 在锁定后可以执行一些计算或传输操作
  // 示例：假设有一个设备指针
  float *deviceData;
  hipMalloc(&deviceData, size * sizeof(float));

  // 将锁定的主机内存内容复制到设备
  err = hipMemcpy(deviceData, hostData, size * sizeof(float),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    std::cerr << "hipMemcpy 失败: " << hipGetErrorString(err) << std::endl;
  }

  // 解锁并释放主机内存
  hipHostUnregister(hostData);
  free(hostData);

  // 释放设备内存
  hipFree(deviceData);

  std::cout << "操作完成。" << std::endl;
  return 0;
}
