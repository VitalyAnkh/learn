#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hip/hip_bf16.h>

__constant__ __hip_bfloat16 a[4];
__constant__ __hip_bfloat16 b[4];

__device__ __hip_bfloat16 test_constant(){
    //a[0]= 2.0;
    // b[0] = {1.0};
    return a[0];
}

__global__ void test_constant_global(){
    test_constant();
    printf("This is arrray a:\n");
    for(int i=0;i<4;i++){
        printf("%f\n",__bfloat162float(a[i]));
    }
    printf("This is array b:\n");
    for(int i=0;i<4;i++){
        printf("%f\n",__bfloat162float(b[i]));
    }
}
int main(){
    __hip_bfloat16* p =(__hip_bfloat16*) malloc(4 *sizeof(__hip_bfloat16));
    for (int i = 0; i < 4; i++)
      p[i] = 2.1;
    hipMemcpyToSymbol(HIP_SYMBOL(a), p,  4*sizeof(__hip_bfloat16));
        test_constant_global<<<1,1>>>();
    hipDeviceSynchronize();

    for (int i = 0; i < 4; i++)
      p[i] = 10.1;

    hipMemcpyToSymbol(HIP_SYMBOL(a), p,  4*sizeof(__hip_bfloat16));
    test_constant_global<<<1,1>>>();
    hipDeviceSynchronize();
}
