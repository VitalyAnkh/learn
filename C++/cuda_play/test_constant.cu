#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hip/hip_bf16.h>

__constant__ __hip_bfloat16 a[4];
__constant__ __hip_bfloat16 b[4];

__device__ __hip_bfloat16 test_constant(){
    a[0]= 2.0;
    // b[0] = {1.0};
    return a[0];
}

__global__ void test_constant_global(){
    test_constant();
    printf("This is arrray a:\n");
    for(int i=0;i<4;i++){
        printf("%f\n",__bfloat162float(a[i]));
    }
    printf("This is array b:\n");
    for(int i=0;i<4;i++){
        printf("%f\n",__bfloat162float(b[i]));
    }
}
int main(){
    test_constant_global<<<1,1>>>();
    hipDeviceSynchronize();
}
